#include "hip/hip_runtime.h"
// Copyright (c) Facebook, Inc. and its affiliates. All Rights Reserved.
#include <torch/types.h>
#include <ATen/ATen.h>
#include <ATen/cuda/Atomic.cuh>
#include <ATen/cuda/HIPContext.h>
#include <ATen/native/cuda/KernelUtils.cuh>

// TODO make it in a common file
#define CUDA_1D_KERNEL_LOOP(i, n)                            \
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; \
       i += blockDim.x * gridDim.x)


template <typename U, typename T>
__device__ T bilinear_interpolate(const U* bottom_data,
    const int height, const int width,
    T y, T x,
    const int index /* index for debug only*/) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    return 0;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int) y;
  int x_low = (int) x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T v1 = bottom_data[y_low * width + x_low];
  T v2 = bottom_data[y_low * width + x_high];
  T v3 = bottom_data[y_high * width + x_low];
  T v4 = bottom_data[y_high * width + x_high];
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  return val;
}

template <typename U, typename T>
__device__ void SingleSampleRoIAlignForward(
    const U* bottom_data, const T spatial_scale, const int height, const int width,  // per level
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois, U* top_data,
    const int n, const int c, const int ph, const int pw
    )
{
  int index = pooled_width * (pooled_height * (n * channels + c) + ph) + pw;
  const T* offset_bottom_rois = bottom_rois + n * 5;
  int roi_batch_ind = offset_bottom_rois[0];

  // Do not using rounding; this implementation detail is critical
  T roi_start_w = offset_bottom_rois[1] * spatial_scale;
  T roi_start_h = offset_bottom_rois[2] * spatial_scale;
  T roi_end_w = offset_bottom_rois[3] * spatial_scale;
  T roi_end_h = offset_bottom_rois[4] * spatial_scale;
  // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
  // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
  // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
  // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

  // Force malformed ROIs to be 1x1
  T roi_width = max(roi_end_w - roi_start_w, (T)1.);
  T roi_height = max(roi_end_h - roi_start_h, (T)1.);
  T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
  T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

  const U* offset_bottom_data = bottom_data + (roi_batch_ind * channels + c) * height * width;

  // We use roi_bin_grid to sample the grid and mimic integral
  int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
  int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

  // We do average (integral) pooling inside a bin
  const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

  T output_val = 0.;
  for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
  {
    const T y = roi_start_h + ph * bin_size_h + static_cast<T>(iy + .5f) * bin_size_h / static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
    for (int ix = 0; ix < roi_bin_grid_w; ix ++)
    {
      const T x = roi_start_w + pw * bin_size_w + static_cast<T>(ix + .5f) * bin_size_w / static_cast<T>(roi_bin_grid_w);

      T val = bilinear_interpolate(offset_bottom_data, height, width, y, x, index);
      output_val += val;
    }
  }
  output_val /= count;

  top_data[index] = output_val;
}

template <typename U>
__device__ void Zero(
    U* top_data,
    const int channels, const int pooled_height, const int pooled_width,
    const int n, const int c, const int ph, const int pw
    )
{
  int index = pooled_width * (pooled_height * (n * channels + c) + ph) + pw;
  top_data[index] = U(0);
}

// rois in math type (float). This is because ROIs come in as float. 
// TODO: Change other blocks producing ROI to support half type as well
template <typename U, typename T>
__global__ void RoIAlignForward(const int nthreads,
    const U* bottom_data, const T spatial_scale, const int height, const int width,  // per-level arguments
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois, U* top_data)
{
  // grid = [n,c,1]
  // block = [ph,pw,1]
  SingleSampleRoIAlignForward(
      bottom_data, spatial_scale, height, width,
      channels, pooled_height, pooled_width, sampling_ratio,
      bottom_rois, top_data,
      blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y
      );
}

template <typename U, typename T, typename index_t>
__global__ void FourLevelsBatchedRoIAlignForward(const int nthreads, const int64_t* counts,
    const U* bottom_data_0, const T spatial_scale_0, const int height_0, const int width_0,  // per-level arguments
    const U* bottom_data_1, const T spatial_scale_1, const int height_1, const int width_1,
    const U* bottom_data_2, const T spatial_scale_2, const int height_2, const int width_2,
    const U* bottom_data_3, const T spatial_scale_3, const int height_3, const int width_3,
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois, U* top_data,
    const index_t* level
    )
{
  // grid = [n,c,1]
  // block = [ph,pw,1]
  int n = blockIdx.x;
  switch (level[n]) {
    case 0:
      SingleSampleRoIAlignForward(
	  bottom_data_0, spatial_scale_0, height_0, width_0,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois, top_data,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
    case 1:
      SingleSampleRoIAlignForward(
	  bottom_data_1, spatial_scale_1, height_1, width_1,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois, top_data,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
    case 2:
      SingleSampleRoIAlignForward(
	  bottom_data_2, spatial_scale_2, height_2, width_2,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois, top_data,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
    case 3:
      SingleSampleRoIAlignForward(
	  bottom_data_3, spatial_scale_3, height_3, width_3,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois, top_data,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
    default:
      Zero(
	  top_data,
	  channels, pooled_height, pooled_width,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
      break;
  }
}

template <typename U, typename T, int nc>
__device__ void bilinear_interpolate_nhwc(
    const U* bottom_data,
    const int height, const int width, const int channels,
    T y, T x,
    T* output_vals) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  int y_low = (int) y;
  int x_low = (int) x;
  int y_high;
  int x_high;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;
  // do bilinear interpolation
  T w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
  int ll = channels * (y_low * width + x_low);
  int lh = channels * (y_low * width + x_high);
  int hl = channels * (y_high * width + x_low);
  int hh = channels * (y_high * width + x_high);
  for (int i = 0;  i < nc;  ++i) {
    T v1 = bottom_data[ll+i];
    T v2 = bottom_data[lh+i];
    T v3 = bottom_data[hl+i];
    T v4 = bottom_data[hh+i];
    T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);
    output_vals[i] += val;
  }
}

template <typename U, typename T, int nc>
__device__ void SingleSampleRoIAlignForwardNHWC(
    const U* bottom_data, const T spatial_scale, const int height, const int width,  // per level
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois, U* top_data,
    const int n, const int ph, const int pw, const int c
    )
{
    int index = channels * (pooled_width * (n * pooled_height + ph) + pw) + c*nc;
    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;
    // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const U* offset_bottom_data = bottom_data + (roi_batch_ind * channels * height * width + c*nc);

    // We use roi_bin_grid to sample the grid and mimic integral
    int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
    int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

    // We do average (integral) pooling inside a bin
    const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

    T output_vals[nc];
    for (int i = 0;  i < nc;  ++i) output_vals[i] = T(0);
    for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
    {
      const T y = roi_start_h + ph * bin_size_h + static_cast<T>(iy + .5f) * bin_size_h / static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
      for (int ix = 0; ix < roi_bin_grid_w; ix ++)
      {
        const T x = roi_start_w + pw * bin_size_w + static_cast<T>(ix + .5f) * bin_size_w / static_cast<T>(roi_bin_grid_w);

        bilinear_interpolate_nhwc<U,T,nc>(offset_bottom_data, height, width, channels, y, x, output_vals);
      }
    }
    U inv_count = U(1) / count;
    for (int i = 0;  i < nc;  ++i) top_data[index+i] = output_vals[i] * inv_count;
}

template <typename U, typename T, int nc>
__device__ void SingleSampleRoIAlignForwardNHWCSR2(
    const U* bottom_data, const T spatial_scale, const int height, const int width,  // per level
    const int channels, const int pooled_height, const int pooled_width,
    const T* bottom_rois, U* top_data,
    const int n, const int ph, const int pw, const int c
    )
{
    int index = channels * (pooled_width * (n * pooled_height + ph) + pw) + c*nc;
    const T* offset_bottom_rois = bottom_rois + n * 5;
    int roi_batch_ind = offset_bottom_rois[0];

    // Do not using rounding; this implementation detail is critical
    T roi_start_w = offset_bottom_rois[1] * spatial_scale;
    T roi_start_h = offset_bottom_rois[2] * spatial_scale;
    T roi_end_w = offset_bottom_rois[3] * spatial_scale;
    T roi_end_h = offset_bottom_rois[4] * spatial_scale;
    // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
    // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
    // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
    // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

    // Force malformed ROIs to be 1x1
    T roi_width = max(roi_end_w - roi_start_w, (T)1.);
    T roi_height = max(roi_end_h - roi_start_h, (T)1.);
    T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
    T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

    const U* offset_bottom_data = bottom_data + (roi_batch_ind * channels * height * width + c*nc);

    // We do average (integral) pooling inside a bin
    const U inv_count = U(0.25);

    T output_vals[nc];
    for (int i = 0;  i < nc;  ++i) output_vals[i] = T(0);
    for (int iy = 0; iy < 2; iy ++) // e.g., iy = 0, 1
    {
      const T y = roi_start_h + ph * bin_size_h + static_cast<T>(iy + .5f) * bin_size_h * T(0.5); // e.g., 0.5, 1.5
      for (int ix = 0; ix < 2; ix ++)
      {
        const T x = roi_start_w + pw * bin_size_w + static_cast<T>(ix + .5f) * bin_size_w * T(0.5);

        bilinear_interpolate_nhwc<U,T,nc>(offset_bottom_data, height, width, channels, y, x, output_vals);
      }
    }
    for (int i = 0;  i < nc;  ++i) top_data[index+i] = output_vals[i] * inv_count;
}

template <typename U, int nc>
__device__ void ZeroNHWC(
    U* top_data,
    const int channels, const int pooled_height, const int pooled_width,
    const int n, const int ph, const int pw, const int c
    )
{
    int index = channels * (pooled_width * (n * pooled_height + ph) + pw) + c*nc;
    for (int i = 0;  i < nc;  ++i) {
      top_data[index+i] = U(0);
    }
}

// rois in math type (float). This is because ROIs come in as float. 
// TODO: Change other blocks producing ROI to support half type as well
template <typename U, typename T>
__global__ void RoIAlignForwardNHWC(const int nthreads,
    const U* bottom_data, const T spatial_scale, const int height, const int width, // per level
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois, U* top_data)
{
  // grid [n,ph,pw]
  // block [c,1,1]
  SingleSampleRoIAlignForwardNHWC<U,T,1>(
      bottom_data, spatial_scale, height, width,
      channels, pooled_height, pooled_width, sampling_ratio,
      bottom_rois, top_data,
      blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x
      );
}

template <typename U, typename T, typename index_t, int nc>
__global__ void FourLevelsBatchedRoIAlignForwardNHWC(const int nthreads, const int64_t* counts,
    const U* bottom_data_0, const T spatial_scale_0, const int height_0, const int width_0,
    const U* bottom_data_1, const T spatial_scale_1, const int height_1, const int width_1,
    const U* bottom_data_2, const T spatial_scale_2, const int height_2, const int width_2,
    const U* bottom_data_3, const T spatial_scale_3, const int height_3, const int width_3,
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois, U* top_data,
    const index_t* level)
{
  // grid [n,ph,pw]
  // block [c,1,1]
  int n = blockIdx.x;
  switch (level[n]) {
    case 0:
      SingleSampleRoIAlignForwardNHWC<U,T,nc>(
	  bottom_data_0, spatial_scale_0, height_0, width_0,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois, top_data,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 1:
      SingleSampleRoIAlignForwardNHWC<U,T,nc>(
	  bottom_data_1, spatial_scale_1, height_1, width_1,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois, top_data,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 2:
      SingleSampleRoIAlignForwardNHWC<U,T,nc>(
	  bottom_data_2, spatial_scale_2, height_2, width_2,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois, top_data,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 3:
      SingleSampleRoIAlignForwardNHWC<U,T,nc>(
	  bottom_data_3, spatial_scale_3, height_3, width_3,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois, top_data,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    default:
      ZeroNHWC<U,nc>(
	  top_data,
	  channels, pooled_height, pooled_width,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
  }
}

template <typename U, typename T, typename index_t, int nc>
__global__ void FourLevelsBatchedRoIAlignForwardNHWCSR2(const int nthreads, const int64_t* counts,
    const U* bottom_data_0, const T spatial_scale_0, const int height_0, const int width_0,
    const U* bottom_data_1, const T spatial_scale_1, const int height_1, const int width_1,
    const U* bottom_data_2, const T spatial_scale_2, const int height_2, const int width_2,
    const U* bottom_data_3, const T spatial_scale_3, const int height_3, const int width_3,
    const int channels, const int pooled_height, const int pooled_width,
    const T* bottom_rois, U* top_data,
    const index_t* level)
{
  // grid [n,ph,pw]
  // block [c,1,1]
  int n = blockIdx.x;
  switch (level[n]) {
    case 0:
      SingleSampleRoIAlignForwardNHWCSR2<U,T,nc>(
	  bottom_data_0, spatial_scale_0, height_0, width_0,
	  channels, pooled_height, pooled_width,
	  bottom_rois, top_data,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 1:
      SingleSampleRoIAlignForwardNHWCSR2<U,T,nc>(
	  bottom_data_1, spatial_scale_1, height_1, width_1,
	  channels, pooled_height, pooled_width,
	  bottom_rois, top_data,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 2:
      SingleSampleRoIAlignForwardNHWCSR2<U,T,nc>(
	  bottom_data_2, spatial_scale_2, height_2, width_2,
	  channels, pooled_height, pooled_width,
	  bottom_rois, top_data,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 3:
      SingleSampleRoIAlignForwardNHWCSR2<U,T,nc>(
	  bottom_data_3, spatial_scale_3, height_3, width_3,
	  channels, pooled_height, pooled_width,
	  bottom_rois, top_data,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    default:
      ZeroNHWC<U,nc>(
	  top_data,
	  channels, pooled_height, pooled_width,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
  }
}

template <typename T>
__device__ void bilinear_interpolate_gradient(
    const int height, const int width,
    T y, T x,
    T & w1, T & w2, T & w3, T & w4,
    int & x_low, int & x_high, int & y_low, int & y_high,
    const int index /* index for debug only*/) {

  // deal with cases that inverse elements are out of feature map boundary
  if (y < -1.0 || y > height || x < -1.0 || x > width) {
    //empty
    w1 = w2 = w3 = w4 = 0.;
    x_low = x_high = y_low = y_high = -1;
    return;
  }

  if (y <= 0) y = 0;
  if (x <= 0) x = 0;

  y_low = (int) y;
  x_low = (int) x;

  if (y_low >= height - 1) {
    y_high = y_low = height - 1;
    y = (T) y_low;
  } else {
    y_high = y_low + 1;
  }

  if (x_low >= width - 1) {
    x_high = x_low = width - 1;
    x = (T) x_low;
  } else {
    x_high = x_low + 1;
  }

  T ly = y - y_low;
  T lx = x - x_low;
  T hy = 1. - ly, hx = 1. - lx;

  // reference in forward
  // T v1 = bottom_data[y_low * width + x_low];
  // T v2 = bottom_data[y_low * width + x_high];
  // T v3 = bottom_data[y_high * width + x_low];
  // T v4 = bottom_data[y_high * width + x_high];
  // T val = (w1 * v1 + w2 * v2 + w3 * v3 + w4 * v4);

  w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;

  return;
}

template <typename U, typename T>
__device__ void SingleSampleRoIAlignBackwardFeature(
    const U* top_diff, 
    const T spatial_scale, const int height, const int width, T* bottom_diff,   // per level
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois,
    const int n, const int c, const int ph, const int pw
    )
{
  int index = pooled_width * (pooled_height * (n * channels + c) + ph) + pw;
  const T* offset_bottom_rois = bottom_rois + n * 5;
  int roi_batch_ind = offset_bottom_rois[0];

  // Do not using rounding; this implementation detail is critical
  T roi_start_w = offset_bottom_rois[1] * spatial_scale;
  T roi_start_h = offset_bottom_rois[2] * spatial_scale;
  T roi_end_w = offset_bottom_rois[3] * spatial_scale;
  T roi_end_h = offset_bottom_rois[4] * spatial_scale;
  // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
  // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
  // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
  // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

  // Force malformed ROIs to be 1x1
  T roi_width = max(roi_end_w - roi_start_w, (T)1.);
  T roi_height = max(roi_end_h - roi_start_h, (T)1.);
  T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
  T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

  T* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels + c) * height * width;

  int top_offset    = (n * channels + c) * pooled_height * pooled_width;
  const U* offset_top_diff = top_diff + top_offset;
  const T top_diff_this_bin = static_cast<T>(offset_top_diff[ph * pooled_width + pw]);

  // We use roi_bin_grid to sample the grid and mimic integral
  int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
  int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

  // We do average (integral) pooling inside a bin
  const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

  for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
  {
    const T y = roi_start_h + ph * bin_size_h + static_cast<T>(iy + .5f) * bin_size_h / static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
    for (int ix = 0; ix < roi_bin_grid_w; ix ++)
    {
      const T x = roi_start_w + pw * bin_size_w + static_cast<T>(ix + .5f) * bin_size_w / static_cast<T>(roi_bin_grid_w);

      T w1, w2, w3, w4;
      int x_low, x_high, y_low, y_high;

      bilinear_interpolate_gradient(height, width, y, x,
	  w1, w2, w3, w4,
	  x_low, x_high, y_low, y_high,
	  index);

      T g1 = top_diff_this_bin * w1 / count;
      T g2 = top_diff_this_bin * w2 / count;
      T g3 = top_diff_this_bin * w3 / count;
      T g4 = top_diff_this_bin * w4 / count;

      if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0)
      {
	gpuAtomicAdd(offset_bottom_diff + y_low * width + x_low, static_cast<T>(g1));
	gpuAtomicAdd(offset_bottom_diff + y_low * width + x_high, static_cast<T>(g2));
	gpuAtomicAdd(offset_bottom_diff + y_high * width + x_low, static_cast<T>(g3));
	gpuAtomicAdd(offset_bottom_diff + y_high * width + x_high, static_cast<T>(g4));
      } // if
    } // ix
  } // iy
}

template <typename U, typename T>
__global__ void RoIAlignBackwardFeature(const int nthreads, const U* top_diff, 
    const T spatial_scale, const int height, const int width, T* bottom_diff,   // per level
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois
    )
{
  // grid = [n,c,1]
  // block = [ph,pw,1]
  SingleSampleRoIAlignBackwardFeature(top_diff,
      spatial_scale, height, width, bottom_diff,
      channels, pooled_height, pooled_width, sampling_ratio,
      bottom_rois,
      blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
} // RoIAlignBackward

template <typename U, typename T, typename index_t>
__global__ void FourLevelsBatchedRoIAlignBackwardFeature(const int nthreads, const int64_t* counts, 
    const U* top_diff,
    const T spatial_scale_0, const int height_0, const int width_0, T* bottom_diff_0,   // per level
    const T spatial_scale_1, const int height_1, const int width_1, T* bottom_diff_1,
    const T spatial_scale_2, const int height_2, const int width_2, T* bottom_diff_2,
    const T spatial_scale_3, const int height_3, const int width_3, T* bottom_diff_3,
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois,
    const index_t* level
    )
{
  // grid = [n,c,1]
  // block = [ph,pw,1]
  int n = blockIdx.x;
  switch (level[n]) {
    case 0:
      SingleSampleRoIAlignBackwardFeature(top_diff,
	  spatial_scale_0, height_0, width_0, bottom_diff_0,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
      break;
    case 1:
      SingleSampleRoIAlignBackwardFeature(top_diff,
	  spatial_scale_1, height_1, width_1, bottom_diff_1,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
      break;
    case 2:
      SingleSampleRoIAlignBackwardFeature(top_diff,
	  spatial_scale_2, height_2, width_2, bottom_diff_2,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
      break;
    case 3:
      SingleSampleRoIAlignBackwardFeature(top_diff,
	  spatial_scale_3, height_3, width_3, bottom_diff_3,
	  channels, pooled_height, pooled_width, sampling_ratio,
	  bottom_rois,
	  n, blockIdx.y, threadIdx.x, threadIdx.y);
      break;
    default:
      break;
  }
}

template <typename U, typename T>
__device__ void SingleSampleRoIAlignBackwardFeatureNHWC(const U* top_diff,
    const T spatial_scale, const int height, const int width, T* bottom_diff,   // per level
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois,
    const int n, const int ph, const int pw, const int c
    )
{
  int index = channels * (pooled_width * (n * pooled_height + ph) + pw) + c;
  const T* offset_bottom_rois = bottom_rois + n * 5;
  int roi_batch_ind = offset_bottom_rois[0];

  // Do not using rounding; this implementation detail is critical
  T roi_start_w = offset_bottom_rois[1] * spatial_scale;
  T roi_start_h = offset_bottom_rois[2] * spatial_scale;
  T roi_end_w = offset_bottom_rois[3] * spatial_scale;
  T roi_end_h = offset_bottom_rois[4] * spatial_scale;
  // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
  // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
  // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
  // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

  // Force malformed ROIs to be 1x1
  T roi_width = max(roi_end_w - roi_start_w, (T)1.);
  T roi_height = max(roi_end_h - roi_start_h, (T)1.);
  T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
  T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

  T* offset_bottom_diff = bottom_diff + (roi_batch_ind * channels * height * width + c);

  int top_offset    = n * channels * pooled_height * pooled_width + c;
  const U* offset_top_diff = top_diff + top_offset;
  const T top_diff_this_bin = static_cast<T>(offset_top_diff[channels * (ph * pooled_width + pw)]);

  // We use roi_bin_grid to sample the grid and mimic integral
  int roi_bin_grid_h = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_height / pooled_height); // e.g., = 2
  int roi_bin_grid_w = (sampling_ratio > 0) ? sampling_ratio : ceil(roi_width / pooled_width);

  // We do average (integral) pooling inside a bin
  const T count = roi_bin_grid_h * roi_bin_grid_w; // e.g. = 4

  for (int iy = 0; iy < roi_bin_grid_h; iy ++) // e.g., iy = 0, 1
  {
    const T y = roi_start_h + ph * bin_size_h + static_cast<T>(iy + .5f) * bin_size_h / static_cast<T>(roi_bin_grid_h); // e.g., 0.5, 1.5
    for (int ix = 0; ix < roi_bin_grid_w; ix ++)
    {
      const T x = roi_start_w + pw * bin_size_w + static_cast<T>(ix + .5f) * bin_size_w / static_cast<T>(roi_bin_grid_w);

      T w1, w2, w3, w4;
      int x_low, x_high, y_low, y_high;

      bilinear_interpolate_gradient(height, width, y, x,
	  w1, w2, w3, w4,
	  x_low, x_high, y_low, y_high,
	  index);

      T g1 = top_diff_this_bin * w1 / count;
      T g2 = top_diff_this_bin * w2 / count;
      T g3 = top_diff_this_bin * w3 / count;
      T g4 = top_diff_this_bin * w4 / count;

      if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0)
      {
	gpuAtomicAdd(offset_bottom_diff + channels * (y_low * width + x_low), static_cast<T>(g1));
	gpuAtomicAdd(offset_bottom_diff + channels * (y_low * width + x_high), static_cast<T>(g2));
	gpuAtomicAdd(offset_bottom_diff + channels * (y_high * width + x_low), static_cast<T>(g3));
	gpuAtomicAdd(offset_bottom_diff + channels * (y_high * width + x_high), static_cast<T>(g4));
      } // if
    } // ix
  } // iy
}

template <typename U, typename T>
__device__ void SingleSampleRoIAlignBackwardFeatureNHWCSR2(const U* top_diff,
    const T spatial_scale, const int height, const int width, T* bottom_diff,   // per level
    const int channels, const int pooled_height, const int pooled_width,
    const T* bottom_rois,
    const int n, const int ph, const int pw, const int c
    )
{
  // grid = [n,ph,pw]
  // block = [c,1,1]

  int index = channels * (pooled_width * (n * pooled_height + ph) + pw) + c;
  const T* offset_bottom_rois = bottom_rois + n * 5;
  int roi_batch_ind = offset_bottom_rois[0];

  // Do not using rounding; this implementation detail is critical
  T roi_start_w = offset_bottom_rois[1] * spatial_scale;
  T roi_start_h = offset_bottom_rois[2] * spatial_scale;
  T roi_end_w = offset_bottom_rois[3] * spatial_scale;
  T roi_end_h = offset_bottom_rois[4] * spatial_scale;
  // T roi_start_w = round(offset_bottom_rois[1] * spatial_scale);
  // T roi_start_h = round(offset_bottom_rois[2] * spatial_scale);
  // T roi_end_w = round(offset_bottom_rois[3] * spatial_scale);
  // T roi_end_h = round(offset_bottom_rois[4] * spatial_scale);

  // Force malformed ROIs to be 1x1
  T roi_width = max(roi_end_w - roi_start_w, (T)1.);
  T roi_height = max(roi_end_h - roi_start_h, (T)1.);
  T bin_size_h = static_cast<T>(roi_height) / static_cast<T>(pooled_height);
  T bin_size_w = static_cast<T>(roi_width) / static_cast<T>(pooled_width);

  const int bottom_diff_numel = channels * height * width;
  T* offset_bottom_diff = bottom_diff + (roi_batch_ind * bottom_diff_numel);

  int top_offset    = n * channels * pooled_height * pooled_width + c;
  const U* offset_top_diff = top_diff + top_offset;
  const T top_diff_this_bin = static_cast<T>(offset_top_diff[channels * (ph * pooled_width + pw)]);

  // We do average (integral) pooling inside a bin
  const T count = 2 * 2; // e.g. = 4

  for (int iy = 0; iy < 2; iy ++) // e.g., iy = 0, 1
  {
    const T y = roi_start_h + ph * bin_size_h + static_cast<T>(iy + .5f) * bin_size_h * T(0.5); // e.g., 0.5, 1.5
    for (int ix = 0; ix < 2; ix ++)
    {
      const T x = roi_start_w + pw * bin_size_w + static_cast<T>(ix + .5f) * bin_size_w * T(0.5);

      T w1, w2, w3, w4;
      int x_low, x_high, y_low, y_high;

      bilinear_interpolate_gradient(height, width, y, x,
	  w1, w2, w3, w4,
	  x_low, x_high, y_low, y_high,
	  index);

      T g1 = top_diff_this_bin * w1 / count;
      T g2 = top_diff_this_bin * w2 / count;
      T g3 = top_diff_this_bin * w3 / count;
      T g4 = top_diff_this_bin * w4 / count;

      if (x_low >= 0 && x_high >= 0 && y_low >= 0 && y_high >= 0)
      {
	at::native::fastAtomicAdd(offset_bottom_diff, channels * (y_low * width + x_low) + c, bottom_diff_numel, static_cast<T>(g1), true);
	at::native::fastAtomicAdd(offset_bottom_diff, channels * (y_low * width + x_high) + c, bottom_diff_numel, static_cast<T>(g2), true);
	at::native::fastAtomicAdd(offset_bottom_diff, channels * (y_high * width + x_low) + c, bottom_diff_numel, static_cast<T>(g3), true);
	at::native::fastAtomicAdd(offset_bottom_diff, channels * (y_high * width + x_high)+ c, bottom_diff_numel, static_cast<T>(g4), true);
      } // if
    } // ix
  } // iy
}

template <typename U, typename T>
__global__ void RoIAlignBackwardFeatureNHWC(const int nthreads, const U* top_diff,
    const T spatial_scale, const int height, const int width, T* bottom_diff,   // per level
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois
    )
{
  // grid [n,ph,pw]
  // block [c,1,1]
  SingleSampleRoIAlignBackwardFeatureNHWC(top_diff,
      spatial_scale,height,width,bottom_diff,
      channels,pooled_height,pooled_width,sampling_ratio,
      bottom_rois,
      blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x);
} // RoIAlignBackward

template <typename U, typename T, typename index_t>
__global__ void FourLevelsBatchedRoIAlignBackwardFeatureNHWC(const int nthreads, const int64_t* counts,
    const U* top_diff,
    const T spatial_scale_0, const int height_0, const int width_0, T* bottom_diff_0,   // per level
    const T spatial_scale_1, const int height_1, const int width_1, T* bottom_diff_1,
    const T spatial_scale_2, const int height_2, const int width_2, T* bottom_diff_2,
    const T spatial_scale_3, const int height_3, const int width_3, T* bottom_diff_3,
    const int channels, const int pooled_height, const int pooled_width, const int sampling_ratio,
    const T* bottom_rois,
    index_t* level
    )
{
  // grid [n,ph,pw]
  // block [c,1,1]
  int n = blockIdx.x;
  switch (level[n]) {
    case 0:
      SingleSampleRoIAlignBackwardFeatureNHWC(top_diff,
	  spatial_scale_0,height_0,width_0,bottom_diff_0,
	  channels,pooled_height,pooled_width,sampling_ratio,
	  bottom_rois,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 1:
      SingleSampleRoIAlignBackwardFeatureNHWC(top_diff,
	  spatial_scale_1,height_1,width_1,bottom_diff_1,
	  channels,pooled_height,pooled_width,sampling_ratio,
	  bottom_rois,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 2:
      SingleSampleRoIAlignBackwardFeatureNHWC(top_diff,
	  spatial_scale_2,height_2,width_2,bottom_diff_2,
	  channels,pooled_height,pooled_width,sampling_ratio,
	  bottom_rois,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 3:
      SingleSampleRoIAlignBackwardFeatureNHWC(top_diff,
	  spatial_scale_3,height_3,width_3,bottom_diff_3,
	  channels,pooled_height,pooled_width,sampling_ratio,
	  bottom_rois,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    default:
      break;
  }
} // RoIAlignBackward

template <typename U, typename T, typename index_t>
__global__ void FourLevelsBatchedRoIAlignBackwardFeatureNHWCSR2(const int nthreads, const int64_t* counts,
    const U* top_diff,
    const T spatial_scale_0, const int height_0, const int width_0, T* bottom_diff_0,   // per level
    const T spatial_scale_1, const int height_1, const int width_1, T* bottom_diff_1,
    const T spatial_scale_2, const int height_2, const int width_2, T* bottom_diff_2,
    const T spatial_scale_3, const int height_3, const int width_3, T* bottom_diff_3,
    const int channels, const int pooled_height, const int pooled_width,
    const T* bottom_rois,
    index_t* level
    )
{
  // grid [n,ph,pw]
  // block [c,1,1]
  int n = blockIdx.x;
  switch (level[n]) {
    case 0:
      SingleSampleRoIAlignBackwardFeatureNHWCSR2(top_diff,
	  spatial_scale_0,height_0,width_0,bottom_diff_0,
	  channels,pooled_height,pooled_width,
	  bottom_rois,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 1:
      SingleSampleRoIAlignBackwardFeatureNHWCSR2(top_diff,
	  spatial_scale_1,height_1,width_1,bottom_diff_1,
	  channels,pooled_height,pooled_width,
	  bottom_rois,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 2:
      SingleSampleRoIAlignBackwardFeatureNHWCSR2(top_diff,
	  spatial_scale_2,height_2,width_2,bottom_diff_2,
	  channels,pooled_height,pooled_width,
	  bottom_rois,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    case 3:
      SingleSampleRoIAlignBackwardFeatureNHWCSR2(top_diff,
	  spatial_scale_3,height_3,width_3,bottom_diff_3,
	  channels,pooled_height,pooled_width,
	  bottom_rois,
	  n, blockIdx.y, blockIdx.z, threadIdx.x);
      break;
    default:
      break;
  }
} // RoIAlignBackward

at::Tensor ROIAlign_forward_cuda(const at::Tensor& input,
                                 const at::Tensor& rois,
                                 const float spatial_scale,
                                 const int pooled_height,
                                 const int pooled_width,
                                 const int sampling_ratio,
				 const bool is_nhwc) {
  AT_ASSERTM(input.is_cuda(), "input must be a CUDA tensor");
  AT_ASSERTM(rois.is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels = is_nhwc ? input.size(3) : input.size(1);
  auto height   = is_nhwc ? input.size(1) : input.size(2);
  auto width    = is_nhwc ? input.size(2) : input.size(3);

  auto output = is_nhwc ? at::empty({num_rois, pooled_height, pooled_width, channels}, input.options()) : 
	  at::empty({num_rois, channels, pooled_height, pooled_width}, input.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (output.numel() == 0) {
    C10_CUDA_CHECK(hipGetLastError());
    return output;
  }
  
  //TODO: Math type is hard coded to float assuming double is not used, if needed, add a case for double as well. 
  //In case of double, it should be <double, double>, not <double, float>
  //TODO: ROIs come in as float, fix other blocks so they come in as same type as input. 
  if (!is_nhwc){
    dim3 grid(num_rois, channels, 1);
    dim3 block(pooled_height, pooled_width, 1);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "ROIAlign_forward", [&] {
      RoIAlignForward<scalar_t, float><<<grid, block, 0, stream>>>(
           output_size,
           input.contiguous().data_ptr<scalar_t>(),
           spatial_scale,
           height,
           width,
           channels,
           pooled_height,
           pooled_width,
           sampling_ratio,
           rois.contiguous().data_ptr<float>(),
           output.data_ptr<scalar_t>());
    });
  }
  else{
    dim3 grid(num_rois, pooled_height, pooled_width);
    dim3 block(channels, 1, 1);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(), "ROIAlign_forward", [&] {
      RoIAlignForwardNHWC<scalar_t, float><<<grid, block, 0, stream>>>(
           output_size,
           input.contiguous().data_ptr<scalar_t>(),
           spatial_scale,
           height,
           width,
           channels,
           pooled_height,
           pooled_width,
           sampling_ratio,
           rois.contiguous().data_ptr<float>(),
           output.data_ptr<scalar_t>());
    });
  }
  C10_CUDA_CHECK(hipGetLastError());
  return output;
}

at::Tensor FourLevelsBatched_ROIAlign_forward_cuda(
		const at::Tensor& input_0,
		const at::Tensor& input_1,
		const at::Tensor& input_2,
		const at::Tensor& input_3,
		const at::Tensor& rois,
		const at::Tensor& rois_counts,
		const at::Tensor& level,
		const float spatial_scale_0,
		const float spatial_scale_1,
		const float spatial_scale_2,
		const float spatial_scale_3,
		const int pooled_height,
		const int pooled_width,
		const int sampling_ratio,
		const bool is_nhwc) {
  AT_ASSERTM(input_0.is_cuda(), "input_0 must be a CUDA tensor");
  AT_ASSERTM(input_1.is_cuda(), "input_1 must be a CUDA tensor");
  AT_ASSERTM(input_2.is_cuda(), "input_2 must be a CUDA tensor");
  AT_ASSERTM(input_3.is_cuda(), "input_3 must be a CUDA tensor");
  AT_ASSERTM(input_0.device() == input_1.device() && input_0.device() == input_2.device() && input_0.device() == input_3.device(), "input_* must all be on same device");
  AT_ASSERTM(input_0.dtype() == input_1.dtype() && input_0.dtype() == input_2.dtype() && input_0.dtype() == input_3.dtype(), "input_* must all be same dtype");
  AT_ASSERTM(rois.is_cuda(), "rois must be a CUDA tensor");
  AT_ASSERTM(rois_counts.is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto channels_0 = is_nhwc ? input_0.size(3) : input_0.size(1);
  auto height_0   = is_nhwc ? input_0.size(1) : input_0.size(2);
  auto width_0    = is_nhwc ? input_0.size(2) : input_0.size(3);
  auto channels_1 = is_nhwc ? input_1.size(3) : input_1.size(1);
  auto height_1   = is_nhwc ? input_1.size(1) : input_1.size(2);
  auto width_1    = is_nhwc ? input_1.size(2) : input_1.size(3);
  auto channels_2 = is_nhwc ? input_2.size(3) : input_2.size(1);
  auto height_2   = is_nhwc ? input_2.size(1) : input_2.size(2);
  auto width_2    = is_nhwc ? input_2.size(2) : input_2.size(3);
  auto channels_3 = is_nhwc ? input_3.size(3) : input_3.size(1);
  auto height_3   = is_nhwc ? input_3.size(1) : input_3.size(2);
  auto width_3    = is_nhwc ? input_3.size(2) : input_3.size(3);
  AT_ASSERT(channels_0 == channels_1 && channels_0 == channels_2 && channels_0 == channels_3, "Channel counts differ");
  auto channels = channels_0;

  auto output = is_nhwc ? at::empty({num_rois, pooled_height, pooled_width, channels}, input_0.options()) : 
	  at::empty({num_rois, channels, pooled_height, pooled_width}, input_0.options());
  auto output_size = num_rois * pooled_height * pooled_width * channels;
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  if (output.numel() == 0) {
    C10_CUDA_CHECK(hipGetLastError());
    return output;
  }

  //TODO: Math type is hard coded to float assuming double is not used, if needed, add a case for double as well. 
  //In case of double, it should be <double, double>, not <double, float>
  //TODO: ROIs come in as float, fix other blocks so they come in as same type as input. 
  if (!is_nhwc){
    dim3 grid(num_rois, channels, 1);
    dim3 block(pooled_height, pooled_width, 1);
    AT_DISPATCH_INDEX_TYPES(level.scalar_type(), "ROIAlign_forward", [&] {
      AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_0.scalar_type(), "ROIAlign_forward", [&] {
        FourLevelsBatchedRoIAlignForward<scalar_t, float, index_t><<<grid, block, 0, stream>>>(
             output_size,
	     rois_counts.numel() == 0 ? NULL : rois_counts.data_ptr<int64_t>(),
             input_0.contiguous().data_ptr<scalar_t>(), spatial_scale_0, height_0, width_0,
             input_1.contiguous().data_ptr<scalar_t>(), spatial_scale_1, height_1, width_1,
             input_2.contiguous().data_ptr<scalar_t>(), spatial_scale_2, height_2, width_2,
             input_3.contiguous().data_ptr<scalar_t>(), spatial_scale_3, height_3, width_3,
             channels,
             pooled_height,
             pooled_width,
             sampling_ratio,
             rois.contiguous().data_ptr<float>(),
             output.data_ptr<scalar_t>(),
  	     level.contiguous().data_ptr<index_t>());
      });
    });
  }
  else{
    const int nc = 4;
    dim3 grid(num_rois, pooled_height, pooled_width);
    if (((channels/nc)*nc) == channels) {
      // launch vectorized kernel if number of channels is divisible by vector size (nc)
      dim3 block(channels/nc, 1, 1);
      AT_DISPATCH_INDEX_TYPES(level.scalar_type(), "ROIAlign_forward", [&] {
	  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_0.scalar_type(), "ROIAlign_forward", [&] {
	      if (sampling_ratio == 2) {
	        FourLevelsBatchedRoIAlignForwardNHWCSR2<scalar_t, float, index_t, nc><<<grid, block, 0, stream>>>(
		    output_size,
		    rois_counts.numel() == 0 ? NULL : rois_counts.data_ptr<int64_t>(),
		    input_0.contiguous().data_ptr<scalar_t>(), spatial_scale_0, height_0, width_0,
		    input_1.contiguous().data_ptr<scalar_t>(), spatial_scale_1, height_1, width_1,
		    input_2.contiguous().data_ptr<scalar_t>(), spatial_scale_2, height_2, width_2,
		    input_3.contiguous().data_ptr<scalar_t>(), spatial_scale_3, height_3, width_3,
		    channels,
		    pooled_height,
		    pooled_width,
		    rois.contiguous().data_ptr<float>(),
		    output.data_ptr<scalar_t>(),
		    level.contiguous().data_ptr<index_t>());
	      } else {
	        FourLevelsBatchedRoIAlignForwardNHWC<scalar_t, float, index_t, nc><<<grid, block, 0, stream>>>(
		    output_size,
		    rois_counts.numel() == 0 ? NULL : rois_counts.data_ptr<int64_t>(),
		    input_0.contiguous().data_ptr<scalar_t>(), spatial_scale_0, height_0, width_0,
		    input_1.contiguous().data_ptr<scalar_t>(), spatial_scale_1, height_1, width_1,
		    input_2.contiguous().data_ptr<scalar_t>(), spatial_scale_2, height_2, width_2,
		    input_3.contiguous().data_ptr<scalar_t>(), spatial_scale_3, height_3, width_3,
		    channels,
		    pooled_height,
		    pooled_width,
		    sampling_ratio,
		    rois.contiguous().data_ptr<float>(),
		    output.data_ptr<scalar_t>(),
		    level.contiguous().data_ptr<index_t>());
	      }
	      });
	  });
    } else {
      // slower fallback option
      dim3 block(channels, 1, 1);
      AT_DISPATCH_INDEX_TYPES(level.scalar_type(), "ROIAlign_forward", [&] {
	  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input_0.scalar_type(), "ROIAlign_forward", [&] {
	      if (sampling_ratio == 2) {
	        FourLevelsBatchedRoIAlignForwardNHWCSR2<scalar_t, float, index_t, 1><<<grid, block, 0, stream>>>(
		    output_size,
		    rois_counts.numel() == 0 ? NULL : rois_counts.data_ptr<int64_t>(),
		    input_0.contiguous().data_ptr<scalar_t>(), spatial_scale_0, height_0, width_0,
		    input_1.contiguous().data_ptr<scalar_t>(), spatial_scale_1, height_1, width_1,
		    input_2.contiguous().data_ptr<scalar_t>(), spatial_scale_2, height_2, width_2,
		    input_3.contiguous().data_ptr<scalar_t>(), spatial_scale_3, height_3, width_3,
		    channels,
		    pooled_height,
		    pooled_width,
		    rois.contiguous().data_ptr<float>(),
		    output.data_ptr<scalar_t>(),
		    level.contiguous().data_ptr<index_t>());
	      } else {
	        FourLevelsBatchedRoIAlignForwardNHWC<scalar_t, float, index_t, 1><<<grid, block, 0, stream>>>(
		    output_size,
		    rois_counts.numel() == 0 ? NULL : rois_counts.data_ptr<int64_t>(),
		    input_0.contiguous().data_ptr<scalar_t>(), spatial_scale_0, height_0, width_0,
		    input_1.contiguous().data_ptr<scalar_t>(), spatial_scale_1, height_1, width_1,
		    input_2.contiguous().data_ptr<scalar_t>(), spatial_scale_2, height_2, width_2,
		    input_3.contiguous().data_ptr<scalar_t>(), spatial_scale_3, height_3, width_3,
		    channels,
		    pooled_height,
		    pooled_width,
		    sampling_ratio,
		    rois.contiguous().data_ptr<float>(),
		    output.data_ptr<scalar_t>(),
		    level.contiguous().data_ptr<index_t>());
	      }
	      });
	  });
    }
  }
  C10_CUDA_CHECK(hipGetLastError());
  return output;
}

// TODO remove the dependency on input and use instead its sizes -> save memory
// NHWC + layout transposes are faster than NCHW, so just keep the NHWC implementation for backward pass
at::Tensor ROIAlign_backward_cuda(const at::Tensor& grad,
                                  const at::Tensor& rois,
                                  const float spatial_scale,
                                  const int pooled_height,
                                  const int pooled_width,
                                  const int batch_size,
                                  const int channels,
                                  const int height,
                                  const int width,
                                  const int sampling_ratio,
				  const bool is_nhwc) {
  AT_ASSERTM(grad.is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(rois.is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto grad_input_options = grad.options().dtype(torch::kFloat32);
  auto grad_input = is_nhwc ? at::zeros({batch_size, height, width, channels}, grad_input_options) : 
	  at::zeros({batch_size, channels, height, width}, grad_input_options);


  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    C10_CUDA_CHECK(hipGetLastError());
    return grad_input;
  }

  //TODO: Math type is hard coded to float assuming double is not used, if needed, add a case for double as well. 
  //In case of double, it should be <double, double>, not <double, float>
  //TODO: ROIs come in as float, fix other blocks so they come in as same type as input. 
  if (!is_nhwc){
    dim3 grid(num_rois, channels, 1);
    dim3 block(pooled_height, pooled_width, 1);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.scalar_type(), "ROIAlign_backward", [&] {
      RoIAlignBackwardFeature<scalar_t, float><<<grid, block, 0, stream>>>(
           grad.numel(),
           grad.contiguous().data_ptr<scalar_t>(),
           spatial_scale,
           height,
           width,
           grad_input.data_ptr<float>(),
           channels,
           pooled_height,
           pooled_width,
           sampling_ratio,
           rois.contiguous().data_ptr<float>());
    });
  }
  else{
    dim3 grid(num_rois, pooled_height, pooled_width);
    dim3 block(channels, 1, 1);
    AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.scalar_type(), "ROIAlign_backward", [&] {
      RoIAlignBackwardFeatureNHWC<scalar_t, float><<<grid, block, 0, stream>>>(
           grad.numel(),
           grad.contiguous().data_ptr<scalar_t>(),
           spatial_scale,
           height,
           width,
           grad_input.data_ptr<float>(),
           channels,
           pooled_height,
           pooled_width,
           sampling_ratio,
           rois.contiguous().data_ptr<float>());
    });
  }
  C10_CUDA_CHECK(hipGetLastError());
  return grad_input;
}

std::vector<at::Tensor> FourLevelsBatched_ROIAlign_backward_cuda(const at::Tensor& grad,
    								 const at::Tensor& grad_counts,
                                                                 const at::Tensor& rois,
                                                                 const at::Tensor& level,
                                                                 const float spatial_scale_0,
                                                                 const float spatial_scale_1,
                                                                 const float spatial_scale_2,
                                                                 const float spatial_scale_3,
                                                                 const int pooled_height,
                                                                 const int pooled_width,
                                                                 const int batch_size,
                                                                 const int channels,
                                                                 const int height_0,
                                                                 const int height_1,
                                                                 const int height_2,
                                                                 const int height_3,
                                                                 const int width_0,
                                                                 const int width_1,
                                                                 const int width_2,
                                                                 const int width_3,
                                                                 const int sampling_ratio,
                                                                 const bool is_nhwc) {
  AT_ASSERTM(grad.is_cuda(), "grad must be a CUDA tensor");
  AT_ASSERTM(grad_counts.is_cuda(), "grad_counts must be a CUDA tensor");
  AT_ASSERTM(rois.is_cuda(), "rois must be a CUDA tensor");

  auto num_rois = rois.size(0);
  auto grad_input_options = grad.options().dtype(torch::kFloat32);
  auto grad_input_0 = is_nhwc ? at::zeros({batch_size, height_0, width_0, channels}, grad_input_options) : at::zeros({batch_size, channels, height_0, width_0}, grad_input_options);
  auto grad_input_1 = is_nhwc ? at::zeros({batch_size, height_1, width_1, channels}, grad_input_options) : at::zeros({batch_size, channels, height_1, width_1}, grad_input_options);
  auto grad_input_2 = is_nhwc ? at::zeros({batch_size, height_2, width_2, channels}, grad_input_options) : at::zeros({batch_size, channels, height_2, width_2}, grad_input_options);
  auto grad_input_3 = is_nhwc ? at::zeros({batch_size, height_3, width_3, channels}, grad_input_options) : at::zeros({batch_size, channels, height_3, width_3}, grad_input_options);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // handle possibly empty gradients
  if (grad.numel() == 0) {
    C10_CUDA_CHECK(hipGetLastError());
    return {grad_input_0, grad_input_1, grad_input_2, grad_input_3};
  }

  //TODO: Math type is hard coded to float assuming double is not used, if needed, add a case for double as well. 
  //In case of double, it should be <double, double>, not <double, float>
  //TODO: ROIs come in as float, fix other blocks so they come in as same type as input. 
  if (!is_nhwc){
    dim3 grid(num_rois, channels, 1);
    dim3 block(pooled_height, pooled_width, 1);
    AT_DISPATCH_INDEX_TYPES(level.scalar_type(), "ROIAlign_backward", [&] {
      AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.scalar_type(), "ROIAlign_backward", [&] {
        FourLevelsBatchedRoIAlignBackwardFeature<scalar_t, float, index_t><<<grid, block, 0, stream>>>(
             grad.numel(),
	     grad_counts.numel() == 0 ? NULL : grad_counts.data_ptr<int64_t>(),
             grad.contiguous().data_ptr<scalar_t>(),
             spatial_scale_0, height_0, width_0, grad_input_0.data_ptr<float>(),
             spatial_scale_1, height_1, width_1, grad_input_1.data_ptr<float>(),
             spatial_scale_2, height_2, width_2, grad_input_2.data_ptr<float>(),
             spatial_scale_3, height_3, width_3, grad_input_3.data_ptr<float>(),
             channels, pooled_height, pooled_width, sampling_ratio,
             rois.contiguous().data_ptr<float>(),
             level.contiguous().data_ptr<index_t>());
      });
    });
  }
  else{
    dim3 grid(num_rois, pooled_height, pooled_width);
    dim3 block(channels, 1, 1);
    AT_DISPATCH_INDEX_TYPES(level.scalar_type(), "ROIAlign_backward", [&] {
      AT_DISPATCH_FLOATING_TYPES_AND_HALF(grad.scalar_type(), "ROIAlign_backward", [&] {
	if (sampling_ratio == 2) {
	  FourLevelsBatchedRoIAlignBackwardFeatureNHWCSR2<scalar_t, float, index_t><<<grid, block, 0, stream>>>(
	      grad.numel(),
	      grad_counts.numel() == 0 ? NULL : grad_counts.data_ptr<int64_t>(),
	      grad.contiguous().data_ptr<scalar_t>(),
	      spatial_scale_0, height_0, width_0, grad_input_0.data_ptr<float>(),
	      spatial_scale_1, height_1, width_1, grad_input_1.data_ptr<float>(),
	      spatial_scale_2, height_2, width_2, grad_input_2.data_ptr<float>(),
	      spatial_scale_3, height_3, width_3, grad_input_3.data_ptr<float>(),
	      channels, pooled_height, pooled_width,
	      rois.contiguous().data_ptr<float>(),
	      level.contiguous().data_ptr<index_t>());
	} else {
	  FourLevelsBatchedRoIAlignBackwardFeatureNHWC<scalar_t, float, index_t><<<grid, block, 0, stream>>>(
	      grad.numel(),
	      grad_counts.numel() == 0 ? NULL : grad_counts.data_ptr<int64_t>(),
	      grad.contiguous().data_ptr<scalar_t>(),
	      spatial_scale_0, height_0, width_0, grad_input_0.data_ptr<float>(),
	      spatial_scale_1, height_1, width_1, grad_input_1.data_ptr<float>(),
	      spatial_scale_2, height_2, width_2, grad_input_2.data_ptr<float>(),
	      spatial_scale_3, height_3, width_3, grad_input_3.data_ptr<float>(),
	      channels, pooled_height, pooled_width, sampling_ratio,
	      rois.contiguous().data_ptr<float>(),
	      level.contiguous().data_ptr<index_t>());
	}
      });
    });
  }
  C10_CUDA_CHECK(hipGetLastError());
  return {grad_input_0, grad_input_1, grad_input_2, grad_input_3};
}
